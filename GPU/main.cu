#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


//cube libraries

//rubik's cube libs
#include "../CPU/cube/parse.c"
#include "../CPU/cube/move.c"

/*ACO parameters*/
	//Number of nodes in the graph
	#define GRAPH_SIZE 1024
	//Number of iteration in ACO algorithm
	#define ACO_ITER_MAX 10
	//evaporation rate
	#define EVAP_RATE 0.3
	//influence rate of the pheroneme
	#define ALPHA 0.8
	//influence rate of the heuristic (distance)
	#define BETA 0.2
	//Initial level of pheroneme
	#define INIT_PHERONEME 5
	//Update pheroneme constant
	#define UPDT_PHERONEME_CONST 2
	//Number of moves allowed through the graph
	#define NSTEPS 2
  //Number of ants
  #define NB_ANT 10
/*End ACO parameters*/

/*GPU parameters*/
	#define BLOCK_SIZE 64
	#define GRID_SIZE 1
	#define ITER_BENCHMARK 100
/*End GPU parameters*/


//function prototypes
float* sum_probability(int* h_graph, float* h_pheroneme, int size);
void update_probability(int* h_graph,float* h_pheroneme,float* h_probability, int size, float* sum)
;
//a macro function that takes as parameters the indexes
//of a 2d matrix and it's row size, and returns the 
//serialized index
#define SERIALIZE(i,j,row_size) i * row_size + j;


//functions prototypes
void datainit_graph(int*, int);
void datainit_pheroneme(float*, int);


__global__ void ACO_kernel(int* d_graph, float* d_pheroneme, float* d_probability, int* d_solutions)
{
  //1) generate a solution (haithem)
  //2) update the pheroneme based on the solution(mohamed)
  int tid = threadIdx.x;

}
  

/*
 * Main program and benchmarking 
 */
int main(int argc, char** argv)
{


  // allocate host memory 
  unsigned int nb_node   = GRAPH_SIZE; 
  unsigned int size      = GRAPH_SIZE*GRAPH_SIZE;             
  unsigned int mem_size  = sizeof(int) * size;     
  int*   h_graph         = (int*)malloc(mem_size); 
  float* h_pheroneme     = (float*)malloc(mem_size);
  float* h_probability   = (float*)malloc(mem_size);
  int*   h_solutions     = (int*)malloc(mem_size); 

  printf("Input size : %d\n", GRAPH_SIZE);
  printf("Grid size  : %d\n", GRID_SIZE);
  printf("Block size : %d\n", BLOCK_SIZE);

  //Initialise the graph, the pheroneme and the probabilities
  datainit_graph(h_graph, nb_node);
  datainit_pheroneme(h_pheroneme, nb_node);
  float* sum =sum_probability(h_graph, h_pheroneme, nb_node);
  update_probability(h_graph, h_pheroneme, h_probability, nb_node, sum);
  
  // allocate device memory
  int* d_graph;
  cutilSafeCall(hipMalloc((void**) &d_graph, mem_size));
  float* d_pheroneme;
  cutilSafeCall(hipMalloc((void**) &d_pheroneme, mem_size));
  float* d_probability;
  cutilSafeCall(hipMalloc((void**) &d_probability, mem_size));
  int* d_solutions;
  cutilSafeCall(hipMalloc((void**) &d_solutions, mem_size));  
  

  // copy host memory to device

  //The graph needs to be copied in the constant memory!!!!!!!!!!!!!!!
  cutilSafeCall(hipMemcpy(d_graph, h_graph, 
				      mem_size, hipMemcpyHostToDevice));

  cutilSafeCall(hipMemcpy(d_pheroneme, h_pheroneme, 
				      mem_size, hipMemcpyHostToDevice));

   cutilSafeCall(hipMemcpy(d_probability, h_probability, 
              mem_size, hipMemcpyHostToDevice));             

  // set up kernel for execution
  printf("Run %d Kernels.\n\n", ITER_BENCHMARK);
  unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
  cutilCheckError(cutStartTimer(timer));  


// execute kernel
  for (int j = 0; j < ITER_BENCHMARK; j++) 
      ACO_kernel<<<GRID_SIZE, BLOCK_SIZE >>>(d_graph, d_pheroneme, d_probability, d_solutions);

  // check if kernel execution generated and error
  cutilCheckMsg("Kernel execution failed");

  // wait for device to finish
  hipDeviceSynchronize();

  // stop and destroy timer
  cutilCheckError(cutStopTimer(timer));
  double dSeconds = cutGetTimerValue(timer)/(1000.0);
  double dNumOps = ITER_BENCHMARK * size;
  double gflops = dNumOps/dSeconds/1.0e9;

  //Log througput
  printf("Throughput = %.4f GFlop/s\n", gflops);
  cutilCheckError(cutDeleteTimer(timer));

  // copy result from device to host
  cutilSafeCall(hipMemcpy(h_solutions, d_solutions, 
				       mem_size, hipMemcpyDeviceToHost));

  // clean up memory
  free(h_graph);
  free(h_pheroneme);
  free(h_probability);
  free(h_solutions);
  cutilSafeCall(hipFree(d_graph));
  cutilSafeCall(hipFree(d_pheroneme));
  cutilSafeCall(hipFree(d_probability));
  cutilSafeCall(hipFree(d_solutions));

  // exit and clean up device status
  hipDeviceReset();
}

// 
void datainit_graph(int* h_graph, int size)
{    
    //same method as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);

            if(i < j) {

                h_graph[index] = 1;
            }
            else {
            h_graph[index] = 0;
          }
        }
    }
 
}

void datainit_pheroneme(float* h_pheroneme, int size)
{
  //same method as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(i < j)
                h_pheroneme[index] = INIT_PHERONEME;
            else{
            h_pheroneme[index] = 0;}
        }
    }

}

void datainit_graph_cube(int *graph,int max_depth) {
    
    //calculate the number of nodes available
    long num_nodes = max_cube_moves(max_depth);

    //calculate the number of nodes that are at depth max_depth -1
    long num_nodes_at_depth_minus_one = max_cube_moves(max_depth - 1);
    int i, j;
    int k =0;
    for (i=0 ; i < num_nodes_at_depth_minus_one; i++) {
      for (j=0 ; j < num_nodes; j++) {

        if ( ( j >= i * 18 + 1  && j< i*18 +1+k ) || ( j>=i*18+1+k+3 && j < (i * 18 + 18 + 1) ) ) {
          graph[i * num_nodes + j] = 1;
        }
        else {
          graph[i * num_nodes + j] = 0;
         }
      }
	if (i != 0 && i%3==0) k+=3;
    }

    //put zeros in the last level of nodes
    for (i=num_nodes_at_depth_minus_one ; i < num_nodes; i++) {
      for (j=0 ; j < num_nodes; j++) {
          graph[i * num_nodes + j] = 0;
      }
    }
}

/*
void datainit_graph_cube(int *graph,int max_depth) {
    
    //calculate the number of nodes available
    long num_nodes = max_cube_moves(max_depth);

    //calculate the number of nodes that are at depth max_depth -1
    long num_nodes_at_depth_minus_one = max_cube_moves(max_depth - 1);
    int i;

    for (i=0 ; i < num_nodes_at_depth_minus_one; i++) {
      if ( i >= i * 18 + 1 && i < (i * 18 + 18) ) {
        graph[i] = 1;
      }
      else {
        graph[i] = 0;
       }
    }

    //put zeros in the last level of nodes
    for (i=num_nodes_at_depth_minus_one ; i < num_nodes; i++) {
      graph[i] = 0;
    }
    int i,j;
    //start from node 2, and keep track of the next node number
    int current_node = 2;


    //initialize the first row
    for (i =0; i < num_nodes; i++) {

          int index = SERIALIZE(i,j,18);
            graph[index] = 0;
            current_node++:      
    }


    for (i =0; i < num_nodes; i++) {

        for (j=0; j<18; j++) {

          int index = SERIALIZE(i,j,18);

          if () { 
            graph[index] = current_node;
            current_node++:
          }
          else {
            graph[index] = 0;
          }

        }
    }   
}
*/



void update_pheroneme(float* h_pheroneme, int size)
{
    int i,j,index;
    //evaporation
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(h_pheroneme[index] != 0)
            {
                h_pheroneme[index] = (1-EVAP_RATE) * h_pheroneme[index];
            }
        }
    }
}


float* sum_probability(int* h_graph, float* h_pheroneme, int size)

{
    int i,j,index;
    float* sum = (float*)malloc(sizeof(float)*size);
    for(i=0 ; i<size ; i++)
    {
        sum[i]=0;
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(h_graph[index] != 0 && h_pheroneme[index] != 0){
                sum[i] += pow(h_pheroneme[index],ALPHA) * pow(1/h_graph[index],BETA);
            }
        }
    }
    return sum;
}


void update_probability(int* h_graph,float* h_pheroneme,float* h_probability, int size, float* sum)
{
    //same methode as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(h_graph[index] != 0 && h_pheroneme[index] != 0)
            {
                h_probability[index] = pow(h_pheroneme[index],ALPHA) * pow(1/h_graph[index],BETA)/sum[i];
            }
            else{
                h_probability[index] = 0;
            }
        }
    }

}

