#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include <hiprand/hiprand_kernel.h>

//cube libraries

//rubik's cube libs
#include "../CPU/cube/parse.c"
#include "../CPU/cube/move.c"

/*ACO parameters*/
  //Number of nodes in the graph
  #define GRAPH_SIZE 1024
  //Number of iteration in ACO algorithm
  #define ACO_ITER_MAX 2
  //evaporation rate
  #define EVAP_RATE 0.3
  //influence rate of the pheromone
  #define ALPHA 0.8
  //influence rate of the heuristic (distance)
  #define BETA 0.2
  //Initial level of pheromone
  #define INIT_PHEROMONE 5
  //Update pheromone constant
  #define UPDT_PHEROMONE_CONST 2
  //Number of ants
  #define NB_ANT 1024
/*End ACO parameters*/

/*GPU parameters*/
  #define GRID_SIZE 1
  #define BLOCK_SIZE 1024
  #define ITER_BENCHMARK 100
/*End GPU parameters*/

//random numbers macros
#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)




//function prototypes
void h_datainit_graph(int*, int);
void h_datainit_pheromone(float*, int);
float* h_sum_probability(int* h_graph, float* h_pheromone, int size);
void h_init_probability(int* h_graph,float* h_pheromone,float* h_probability, int size, float* h_sum);
int* h_find_best_solution(int* h_solutions, int* h_length, int size);

//a macro function that takes as parameters the indexes
//of a 2d matrix and it's row size, and returns the 
//serialized index
#define SERIALIZE(i,j,row_size) i * row_size + j;

__global__ void init_d_solutions(int *d_solutions) {

  int tid = threadIdx.x;
  int index,j;

  //initialize the array that contain the solution
  //each thread initialise one row
  for(j=0; j<GRAPH_SIZE ; j++)
  {
    index = SERIALIZE(tid,j,GRAPH_SIZE);
    d_solutions[index]=0;
  }

}


__global__ void generate_solutions(float* d_probability,float* d_random_numbers,int* d_solutions) {

  int tid = threadIdx.x;
  int index;
  
  // //Generate the solution
  float rdm;
  
  index=SERIALIZE(tid,1,GRAPH_SIZE);
  //For the cube it is going to be loop until NB_STEP is reached or solution found 
  while(d_solutions[index-1] != GRAPH_SIZE-1)
  {
      //select the next node based on the probability
      //take a random number between 0 and 1 with 0 excluded
      rdm=d_random_numbers[index];

      //Probability to select the next node
      float Pnext = 0;

      int j,ip;
      for(j=0; j<GRAPH_SIZE; j++)
      {
          ip = SERIALIZE(d_solutions[index-1], j, GRAPH_SIZE);
          Pnext += d_probability[ip];

          //if the random number is less or equal to
          //the probability to select the next node we select it
          if( rdm <= Pnext )
          {
              d_solutions[index]=j;
              break;
          }
       }

      index++;
  }
}

__global__ void ACO_kernel(int* d_graph, float* d_pheromone, float* d_probability, float* d_random_numbers, int* d_solutions,int* d_length)
{

  int tid = threadIdx.x;
  int index,j;
  

  //Calculate the length of the path for each ant
  d_length[tid]=0;
  j=0;
  index=SERIALIZE(tid,j,GRAPH_SIZE)
  while(d_solutions[index] != GRAPH_SIZE-1)
  {
      d_length[tid] += d_graph[d_solutions[index]*GRAPH_SIZE + d_solutions[index+1]];
      j++;
      index=SERIALIZE(tid,j,GRAPH_SIZE);
  }

  //Update the pheromone based on constructed solution
  //Each ant update its own path in the pheromone matrix
  index=SERIALIZE(tid,0,GRAPH_SIZE);
  while(d_solutions[index] != GRAPH_SIZE-1)
    {
        j=SERIALIZE(d_solutions[index],d_solutions[index+1],GRAPH_SIZE);
        d_pheromone[j] += UPDT_PHEROMONE_CONST/d_length[tid];
        index++;
    }

}

__global__ void update_pheromone_kernel(float* d_pheromone)
{
  int tid = threadIdx.x;
  int index;
  //pheromone evaporation
  for(int j=0; j<GRAPH_SIZE; j++)
  {
    index=SERIALIZE(tid,j,GRAPH_SIZE);
    d_pheromone[index] = (1-EVAP_RATE) * d_pheromone[index];
  }
}


__global__ void update_probability_kernel1(int* d_graph, float* d_pheromone, float* d_probability)
{
  int tid = threadIdx.x;
  int index;

  //update probability based on the new pheromone matrix
  for(int j=0; j<GRAPH_SIZE; j++)
  {
      index = SERIALIZE(tid,j,GRAPH_SIZE);
      if(d_graph[index] != 0)
        {
           d_probability[index] = pow((double)d_pheromone[index],ALPHA) * pow( 1/(double)d_graph[index], BETA );
        }
  }

}

__device__ float d_f(float x, float y) { return x + y; }

__global__ void sum_probablity_kernel(float* d_probability, float* d_sum)
{
  int tid = threadIdx.x;

  //Use the reduce algorithm we did on the lab to calculate the sum of probability
  //Each thread will reduce each row of the probability matrix
  float tmp1,tmp2;
  int nsteps;
  int fi,si;
  int index;
  for(int j=0; j<GRAPH_SIZE; j++)
  {

    index=SERIALIZE(j, tid, GRAPH_SIZE);

    fi = SERIALIZE(j, tid * 2, GRAPH_SIZE);
    si = SERIALIZE(j, tid * 2 + 1, GRAPH_SIZE);
    
    d_sum[index] = d_f(d_probability[fi] , d_probability[si]);
    __syncthreads();

    nsteps = log2((float)GRAPH_SIZE);

     #pragma unroll
     for (int k=1; k < nsteps; k++) {
        
     //first read all elements, then write
     //this is to avoid race condition ...
     //if one thread writes before another, the result will be wrong
          tmp1 =  d_sum[fi];
          tmp2 =  d_sum[si];
          __syncthreads();
          d_sum[index] = d_f(tmp1, tmp2);
    
    __syncthreads();    
    }
  }
}


__global__ void update_probability_kernel2(float* d_probability,float* d_sum)
{
  int tid = threadIdx.x;
  int index;

  //update probability based on the new pheromone matrix
  int index_sum;
  for(int j=0; j<GRAPH_SIZE; j++)
  {
      index = SERIALIZE(tid,j,GRAPH_SIZE);
      index_sum = SERIALIZE(tid,0,GRAPH_SIZE);
      d_probability[index] /= d_sum[index_sum];
  }

}

/*
 * Main program and benchmarking 
 */
int main(int argc, char** argv)
{


  // allocate host memory 
  unsigned int nb_node              = GRAPH_SIZE; 
  unsigned int size_graph           = GRAPH_SIZE*GRAPH_SIZE;
  unsigned int mem_size_graph_int   = sizeof(int) * size_graph;
  unsigned int mem_size_graph_float = sizeof(float) * size_graph;
  unsigned int mem_size_ant         = sizeof(int) * NB_ANT;
  unsigned int mem_size_solution    = sizeof(int)*NB_ANT*GRAPH_SIZE;    
  int*   h_graph                    = (int*)malloc(mem_size_graph_int); 
  float* h_pheromone                = (float*)malloc(mem_size_graph_float);
  float* h_probability              = (float*)malloc(mem_size_graph_float);
  int*   h_solutions                = (int*)malloc(mem_size_solution);
  int*   h_length                   = (int*)malloc(mem_size_ant);

  //Initialise random numbers
  float *d_random_numbers;
  //create hiprand generator object
  hiprandGenerator_t gen;
  /* Allocate n floats on device */
  CUDA_CALL(hipMalloc((void **)&d_random_numbers, NB_ANT * nb_node *sizeof(float)));

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, 
              HIPRAND_RNG_PSEUDO_DEFAULT));
  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
              time(NULL)));
  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniform(gen, d_random_numbers, NB_ANT * nb_node ));



  printf("Input size : %d\n", GRAPH_SIZE);
  printf("Grid  size : %d\n", GRID_SIZE);
  printf("Block size : %d\n", BLOCK_SIZE);

  //Initialise the graph, the pheromone and the probabilities
  h_datainit_graph(h_graph, nb_node);
  h_datainit_pheromone(h_pheromone, nb_node);
  float* h_sum = h_sum_probability(h_graph, h_pheromone, nb_node);
  h_init_probability(h_graph, h_pheromone, h_probability, nb_node, h_sum);


  // allocate device memory
  int* d_graph;
  cutilSafeCall(hipMalloc((void**) &d_graph, mem_size_graph_int));
  float* d_pheromone;
  cutilSafeCall(hipMalloc((void**) &d_pheromone, mem_size_graph_float));
  float* d_probability;
  cutilSafeCall(hipMalloc((void**) &d_probability, mem_size_graph_float));
  int* d_solutions;
  cutilSafeCall(hipMalloc((void**) &d_solutions, mem_size_solution));

  //Array that contain the length of the path generated by each ant
  int* d_length;
  cutilSafeCall(hipMalloc((void**) &d_length, mem_size_ant));

  //Array that contain the sum of the probability
  float* d_sum;
  cutilSafeCall(hipMalloc((void**) &d_sum, mem_size_graph_float));  
  

  // copy host memory to device

  //The graph needs to be copied in the constant memory!!!!!!!!!!!!!!!
  cutilSafeCall(hipMemcpy(d_graph, h_graph, 
              mem_size_graph_int, hipMemcpyHostToDevice));

  cutilSafeCall(hipMemcpy(d_pheromone, h_pheromone, 
              mem_size_graph_float, hipMemcpyHostToDevice));

  cutilSafeCall(hipMemcpy(d_probability, h_probability, 
              mem_size_graph_float, hipMemcpyHostToDevice));             

  // set up kernel for execution
  printf("Run %d Kernels.\n\n", ITER_BENCHMARK);
  unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
  cutilCheckError(cutStartTimer(timer));  

int* h_best_solution;
// execute kernel
  //for (int j = 0; j < ITER_BENCHMARK; j++) 
      for(int i = 0; i < ACO_ITER_MAX; i++){

        init_d_solutions<<<1, NB_ANT >>>( d_solutions);
        generate_solutions<<<1, NB_ANT>>>(d_probability,d_random_numbers,d_solutions);
        ACO_kernel<<<1, NB_ANT >>>(d_graph, d_pheromone, d_probability, d_random_numbers, d_solutions, d_length);
        update_pheromone_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_pheromone);
        update_probability_kernel1<<<GRID_SIZE,BLOCK_SIZE>>>(d_graph, d_pheromone, d_probability);
        sum_probablity_kernel<<<GRID_SIZE,BLOCK_SIZE/2>>>(d_probability,d_sum);
        update_probability_kernel2<<<GRID_SIZE,BLOCK_SIZE-1>>>(d_probability,d_sum);

        // copy result from device to host
        cutilSafeCall(hipMemcpy(h_solutions, d_solutions, 
            mem_size_solution, hipMemcpyDeviceToHost));

        cutilSafeCall(hipMemcpy(h_length, d_length, 
           mem_size_ant, hipMemcpyDeviceToHost));

        //find the best solution and its length
        h_best_solution = h_find_best_solution(h_solutions,h_length,NB_ANT);



        //regenerate random numbers
        /* Set seed */
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                    time(NULL)));
        /* Generate n floats on device */
        CURAND_CALL(hiprandGenerateUniform(gen, d_random_numbers, NB_ANT * nb_node ));
     }


  printf("the best path is: \n");
  int i = 1;
  printf("%d ",h_best_solution[0]);
  while(h_best_solution[i-1] != GRAPH_SIZE-1)
  {
    printf("%d ",h_best_solution[i]);
    i++;
  }
   printf("\n");


  // printf("last set of solutions \n");
  // int index;
  // for(int i=0; i<NB_ANT; i++)
  // {
  //   for(int j=0; j<nb_node; j++)
  //   {
  //       index = SERIALIZE(i,j,nb_node);
  //       printf("%d ",h_solutions[index]);
  //   }
  //    printf("\n");
  // }

  // printf("last set of length solution \n");
  // for(int i=0; i<NB_ANT; i++)
  // {
  //   printf("%d ",h_length[i]);
  // }
  
  //  printf("\n");

  // check if kernel execution generated and error
  cutilCheckMsg("Kernel execution failed");

  // wait for device to finish
  hipDeviceSynchronize();

  // stop and destroy timer
  cutilCheckError(cutStopTimer(timer));
  double dSeconds = cutGetTimerValue(timer)/(1000.0);
  double dNumOps = (size_graph * 4 + NB_ANT * (2*GRAPH_SIZE + 1));
  double gflops = dNumOps/dSeconds/1.0e9;

  //Log througput
  printf("Throughput = %.4f GFlop/s\n", gflops);
  printf("Times = %.4f s\n", dSeconds);
  cutilCheckError(cutDeleteTimer(timer));

  // clean up memory
  free(h_graph);
  free(h_pheromone);
  free(h_probability);
  free(h_solutions);
  free(h_length);
  free(h_sum);
  free(h_best_solution);
  cutilSafeCall(hipFree(d_graph));
  cutilSafeCall(hipFree(d_pheromone));
  cutilSafeCall(hipFree(d_probability));
  cutilSafeCall(hipFree(d_solutions));
  cutilSafeCall(hipFree(d_length));
  cutilSafeCall(hipFree(d_sum));

  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_CALL(hipFree(d_random_numbers)); 

  // exit and clean up device status
  hipDeviceReset();
}


void h_datainit_graph(int* h_graph, int size)
{    
    //same method as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);

            if(i < j) {

                h_graph[index] = 1;
            }
            else {
            h_graph[index] = 0;
          }
        }
    }
 
}

void h_datainit_pheromone(float* h_pheromone, int size)
{
  //same method as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(i < j)
            {
              h_pheromone[index] = INIT_PHEROMONE;
            }
            else{
            h_pheromone[index] = 0;
          }
        }
    }

}


void h_datainit_graph_cube(int * h_graph,int max_depth) {
    
    int i, j;

    //calculate the number of nodes available
    long num_nodes = max_cube_moves(max_depth);

    //calculate the number of nodes that are at depth max_depth -1
    long num_nodes_at_depth_minus_one = max_cube_moves(max_depth - 1);


    //let's initialize the first row separately because it doesn't follow the trend
    for (j=1 ; j < num_nodes; j++) {
    
        int index = SERIALIZE(0,j,num_nodes);
        if (j < 19 ) {
          h_graph[index] = 1;
        }
        else {
          h_graph[index] = 0;
        }
      }
    
    // we do the rest of the rows that contain "1's"
    // Since it's a tree, we shift to the right in each row
    //because the nodes are only connected to only 15 other nodes, 
      //we skip three nodes in each row because after doing a move,
      //we don't want to do a same-face rotation again
      // 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 
      // 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 0 0 0 0 0 0 0 0
      // 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 0 0 0 0 0 0 0 0
      // 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 0 0 0 0 0 0
      // 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 1 1 1 0 0 0 1 1 1 1 1 1 1 1 1 1 1 1 0 0 0 0 0 0 0 0
      //....
    for (i=1 ; i < num_nodes_at_depth_minus_one; i++) {
      for (j=0 ; j < num_nodes; j++) {

        int index = SERIALIZE(i,j,num_nodes);

        if (  ( j >= 18 + (i-1) * 15 + 1) &&   j < (18 + (i-1) * 15 + 15 + 1     ) )  {

          h_graph[index] = 1;
        
        }
        else {
          h_graph[index] = 0;
         }
      }

    }

    //put zeros in the last level of nodes
    for (i=num_nodes_at_depth_minus_one ; i < num_nodes; i++) {
      for (j=0 ; j < num_nodes; j++) {
          h_graph[i * num_nodes + j] = 0;
      }
    }
}

/*
void datainit_graph_cube(int *graph,int max_depth) {
    
    //calculate the number of nodes available
    long num_nodes = max_cube_moves(max_depth);

    //calculate the number of nodes that are at depth max_depth -1
    long num_nodes_at_depth_minus_one = max_cube_moves(max_depth - 1);
    int i;

    for (i=0 ; i < num_nodes_at_depth_minus_one; i++) {
      if ( i >= i * 18 + 1 && i < (i * 18 + 18) ) {
        graph[i] = 1;
      }
      else {
        graph[i] = 0;
       }
    }

    //put zeros in the last level of nodes
    for (i=num_nodes_at_depth_minus_one ; i < num_nodes; i++) {
      graph[i] = 0;
    }
    int i,j;
    //start from node 2, and keep track of the next node number
    int current_node = 2;


    //initialize the first row
    for (i =0; i < num_nodes; i++) {

          int index = SERIALIZE(i,j,18);
            graph[index] = 0;
            current_node++:      
    }


    for (i =0; i < num_nodes; i++) {

        for (j=0; j<18; j++) {

          int index = SERIALIZE(i,j,18);

          if () { 
            graph[index] = current_node;
            current_node++:
          }
          else {
            graph[index] = 0;
          }

        }
    }   
}
*/

float* h_sum_probability(int* h_graph, float* h_pheromone, int size)

{
    int i,j,index;
    float* sum = (float*)malloc(sizeof(float)*size);
    for(i=0 ; i<size ; i++)
    {
        sum[i]=0;
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(h_graph[index] != 0){
                sum[i] += pow(h_pheromone[index],ALPHA) * pow(1/h_graph[index],BETA);
            }
        }
    }
    return sum;
}


void h_init_probability(int* h_graph,float* h_pheromone,float* h_probability, int size, float* h_sum)
{
    //same methode as the CPU version
    int i,j,index;
    for(i=0 ; i<size ; i++)
    {
        for(j=0 ; j<size ; j++)
        {
            index = SERIALIZE(i,j,size);
            if(h_graph[index] != 0)
            {
                h_probability[index] = pow(h_pheromone[index],ALPHA) * pow(1/h_graph[index],BETA)/h_sum[i];
            }
            else{
                h_probability[index] = 0;
            }
        }
    }

}
//
int* h_find_best_solution(int* h_solutions, int* h_length, int size)
{
  //find the shortest length and path
  int* h_best_solution = (int*)malloc(sizeof(int) * GRAPH_SIZE);
  int Lmin=h_length[0];
  int index;
  for(int i=1; i<size; i++)
  {
      if(h_length[i]<=Lmin)
      {
        Lmin = h_length[i];
        index = SERIALIZE(i,0,GRAPH_SIZE);
        memcpy(h_best_solution, &(h_solutions[index]), sizeof(int)*GRAPH_SIZE);
      }   
  }
  return h_best_solution;
}
